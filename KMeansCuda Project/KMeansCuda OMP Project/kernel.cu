#include "hip/hip_runtime.h"
#include "kernel.h"
#include "Point.h"
#include <stdio.h>

const int MAX_THREADS_PER_BLOCK = 1024;
int numOfBlocks;
int dev_numOfPoints;
int chunkSize;
Point* dev_allPoints;
float* dev_distancesArr;

__global__ void setPointsForTimeIncrement(Point* dev_allPoints, float cosCalc, float sinCalc, int numOfPoints)
{
	//int currentThread = threadIdx.x;
	//int currentBlock = blockIdx.x;
	////setting the position of the thread.
	//int pos = currentBlock*MAX_THREADS_PER_BLOCK + currentThread;
	//if(pos < numOfPoints)
	//{
	//	dev_allPoints[pos].x = (float)dev_allPoints[pos].a + (dev_allPoints[pos].radius*cosCalc);
	//	dev_allPoints[pos].y = (float)dev_allPoints[pos].b + (dev_allPoints[pos].radius*sinCalc);
	//}
}

void setPointsDevice(Point* allPoints, float timeIncrement, float timeInterval)
{
	hipError_t cudaStatus;
	//set the cos/sin calc for the threads ONE TIME ONLY.
	float cosCalc = (float)cos((2*acos(-1.0) *timeIncrement)/timeInterval);
	float sinCalc = (float)sin((2*acos(-1.0) *timeIncrement)/timeInterval);

	setPointsForTimeIncrement<<<numOfBlocks, MAX_THREADS_PER_BLOCK>>>(dev_allPoints, cosCalc, sinCalc, dev_numOfPoints);
	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "setPointsForTimeIncrement launch failed: %s\n", hipGetErrorString(cudaStatus));
		releaseDeviceMemory();
		return;
	}

	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaDevice Synchronize returned error code %d\n", cudaStatus);
		releaseDeviceMemory();
		return;
	}

	cudaStatus = hipMemcpy(allPoints, dev_allPoints, dev_numOfPoints*sizeof(Point), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy from device to host failed!");
		releaseDeviceMemory();
		return;
	}

	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaDevice Synchronize returned error code %d\n", cudaStatus);
		releaseDeviceMemory();
		return;
	}
}

void copyPointsToDevice(Point* allPoints, int numOfPoints, int chunk)
{
	hipError_t cudaStatus;
	dev_numOfPoints = numOfPoints;
	chunkSize = chunk;

	cudaStatus = hipMalloc((void**)&dev_allPoints, dev_numOfPoints*sizeof(Point));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc for dev_allPoints failed");
		releaseDeviceMemory();
		return;
	}
	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaDevice Synchronize returned error code %d\n", cudaStatus);
		releaseDeviceMemory();
		return;
	}
	cudaStatus = hipMemcpy(dev_allPoints, allPoints, dev_numOfPoints*sizeof(Point), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy from host to device failed!");
		releaseDeviceMemory();
		return;
	}

	numOfBlocks = dev_numOfPoints/MAX_THREADS_PER_BLOCK;
	if(dev_numOfPoints%MAX_THREADS_PER_BLOCK != 0)
		numOfBlocks += 1;

	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaDevice Synchronize returned error code %d\n", cudaStatus);
		releaseDeviceMemory();
		return;
	}
}

void initDevice()
{
	hipError_t cudaStatus;
//	hipDeviceProp_t deviceProp;

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!");
		releaseDeviceMemory();
		return;
	}

	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaDevice Synchronize returned error code %d\n", cudaStatus);
		releaseDeviceMemory();
		return;
	}

//	hipGetDeviceProperties(&deviceProp, 0);
//	maxNumberOfThreadPerBlock = deviceProp.maxThreadsPerBlock;
}

void releaseDeviceMemory()
{
	hipFree(dev_allPoints);
}