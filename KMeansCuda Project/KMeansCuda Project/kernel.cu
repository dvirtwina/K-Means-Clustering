#include "hip/hip_runtime.h"
#include "kernel.h"
#include "Point.h"
#include <stdio.h>
#include <time.h>
using namespace std;

const int MAX_THREADS_PER_BLOCK = 1024;
int numOfBlocks;
int dev_numOfPoints;
Point* dev_allPoints;
float* dev_distancesArr;

/** each thread in the GPU executes a distance measurment from the given point to the thread's assigned point.*/
__global__ void setDistancesForPoint(Point* dev_allPoints, int dev_numOfPoints, float* dev_distancesArr, float pntX, float pntY)
{
	float x,y, dist;
	int currentThread = threadIdx.x;
	int currentBlock = blockIdx.x;
	//set position of the thread.
	int pos = currentBlock*MAX_THREADS_PER_BLOCK + currentThread;

	if(pos < dev_numOfPoints)
	{
		x = (pntX - dev_allPoints[pos].x)*(pntX - dev_allPoints[pos].x);
		y = (pntY - dev_allPoints[pos].y)*(pntY - dev_allPoints[pos].y);
		dist = sqrt(x + y);
		dev_distancesArr[pos] = dist;
	}
}

/**calculate the distance from a given point to every other point in the array.*/
void cudaDistancesOfPoint(Point* allPoints, float* distances, float pntX, float pntY)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_distancesArr, dev_numOfPoints*sizeof(float));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc for dev_distancesArr failed\n");
		releaseDeviceMemory();
		return;
	}
	setDistancesForPoint<<<numOfBlocks, MAX_THREADS_PER_BLOCK>>>(dev_allPoints, dev_numOfPoints, dev_distancesArr, pntX, pntY);

	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "setDistancesForPoint launch failed: %s\n", hipGetErrorString(cudaStatus));
		releaseDeviceMemory();
		return;
	}

	cudaStatus = hipDeviceSynchronize(); 
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaDevice Synchronize returned error code %d\n", cudaStatus);
		releaseDeviceMemory();
		return;
	}

	cudaStatus = hipMemcpy(distances, dev_distancesArr, dev_numOfPoints*sizeof(float), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy from device to host failed!\n");
		releaseDeviceMemory();
		return;
	}

	hipFree(dev_distancesArr);
}

/** Copy an array of points to the GPU device.*/
void copyPointsToDevice(Point* allPoints, int numOfPoints)
{
	hipError_t cudaStatus;
	dev_numOfPoints = numOfPoints;

	cudaStatus = hipMalloc((void**)&dev_allPoints, dev_numOfPoints*sizeof(Point));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc for dev_allPoints failed\n");
		releaseDeviceMemory();
		return;
	}

	cudaStatus = hipMemcpy(dev_allPoints, allPoints, dev_numOfPoints*sizeof(Point), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy from host to device failed!\n");
		releaseDeviceMemory();
		return;
	}

	numOfBlocks = dev_numOfPoints/MAX_THREADS_PER_BLOCK;
	if(dev_numOfPoints%MAX_THREADS_PER_BLOCK != 0)
		numOfBlocks += 1;
}

/** Instantiating GPU device.*/
void initDevice()
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!\n");
		releaseDeviceMemory();
		return;
	}
}

void releaseDeviceMemory()
{
	hipFree(dev_allPoints);
}